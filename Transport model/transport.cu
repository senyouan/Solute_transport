#include "hip/hip_runtime.h"

#define _CRT_SECURE_NO_WARNINGS
#include "global.h"
#include "initialize.cu"
#include "Output_PNM.cu"
#include "Iteration_PNM.cu"

int main(int argc, char *argv[])
{
	void setup();
	void Iteration();
	void initialize();

    setup();
    initialize();
 
    clock_t start, finish; 
	start = clock();
	//ieration
	double outT = 1.0; //double outT = 1.0;
	//t_sum = 335369.691918;
	for (t = 0; t < Time; t++)
	{
	    Iteration();
		t_sum = t_sum + hmint;
        if (int(pow(10, outT)) == t)
	    {
			Output_PNM();
			output(t);
			outT = outT + 0.1;
		}
        printf("%d\n", t);
		//getchar();
		finish=clock();
		if ((finish-start)/CLOCKS_PER_SEC > 47.5 * 3600)
    	{
	        break;
    	}
	}
    finish=clock();
    std::cout <<"total time for ieration:" << (double)(finish-start)/CLOCKS_PER_SEC << "\n";
    Output_PNM();
    Free_PNM();
	output(t);
}

//------------------------------------------------------------------------------
void setup()
{
    int i;
	//-----------------------------------------------------------
	sprintf(filename, "pth2pb_full.txt");
	if ((fp = fopen(filename, "r")) == NULL){ printf("Reading pth2pb_full.txt error.\n"); getchar(); exit(1); }
	for (i = 0; i < TF; i++)
	{
		fscanf(fp, "%d", &Prr[i]);
		fscanf(fp, "%lf", &Trans[i]);
		fscanf(fp, "%lf", &Flux[i]);
	}
	fclose(fp);
	//-----------------------------------------------------------
	sprintf(filename, "Volume.txt");
	if ((fp = fopen(filename, "r")) == NULL){ printf("Reading Volume.txt error.\n"); getchar(); exit(1); }
	for (i = 0; i < TF; i++)
	{
		fscanf(fp, "%lf", &Volume[i]);
	}
	fclose(fp);
	//-----------------------------------------------------------
	sprintf(filename, "coord_nr.txt");
	if ((fp = fopen(filename, "r")) == NULL){ printf("Reading coord_nr.txt error.\n"); getchar(); exit(1); }
	for (i = 0; i < PE; i++)
	{
		fscanf(fp, "%d", &CoordMat[i]);
	}
	fclose(fp);
	//-----------------------------------------------------------
	sprintf(filename, "coord_nr_sum.txt");
	if ((fp = fopen(filename, "r")) == NULL){ printf("Reading coord_nr_sum.txt error.\n"); getchar(); exit(1); }
	for (i = 0; i < PE; i++)
	{
		fscanf(fp, "%d", &CoordMat_sum[i]);
	}
	fclose(fp);

	//-----------------------------------------------------------
	//initialize the consentration
	Cons = new double [PE + 2];
	for(i = 0; i < PE; i++)
    {
        Cons[i] = A0;
    }
    for(i = 0; i < TP; i++)
    {
        Cons[i] = Ai;
    }
	Cons[PE] = Ai;//inlet
	Cons[PE + 1] = A0; //outlet}

	// for continue
	/*sprintf(filename, "Concentration_203235701.txt");
	if ((fp = fopen(filename, "r")) == NULL){ printf("Reading coord_nr_sum.txt error.\n"); getchar(); exit(1); }
	fscanf(fp, "%lf", &t_sum);
	for (i = 0; i < PE; i++)
	{
		fscanf(fp, "%lf", &Cons[i]);
	}
	fclose(fp);*/

}
void output(int t)
{
    int i;
    sprintf(filename, "Concentration_%d.txt",t);
    if ((fp = fopen(filename, "w")) == NULL) { printf("Concentration.txt open error.\n"); getchar(); exit(1); }
	fprintf(fp, "%.15f\n", t_sum);
    for (i = 0; i < PE; i++)
    {
        fprintf(fp, "%.15f\n", Cons[i]);
    }
    fclose(fp);
}
