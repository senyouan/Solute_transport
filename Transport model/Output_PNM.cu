//#include "global.h"

hipError_t Output_PNM() // Copy GPU data to CPU
{
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);

    cudaStatus = hipMemcpy(Cons, D_Cons, (PE + 2) * sizeof(double), hipMemcpyDeviceToHost);
	//cudaStatus = hipMemcpy(Cons, D_deltaCons, (PE) * sizeof(double), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Copy data failed, Output!");
	}
	return cudaStatus;
}



hipError_t Free_PNM()  // Free memory in GPU

{
	hipError_t cudaStatus = hipSuccess;
	
	hipFree(D_Flux);
	hipFree(D_Trans);
	hipFree(D_Cons);
	hipFree(D_Volume);
	hipFree(D_mintime);
	hipFree(D_deltaCons);
	hipFree(D_CoordMat);
	hipFree(D_CoordMat_sum);
	hipFree(D_Prr);
	hipFree(d_minindex);
    return cudaStatus;
}
