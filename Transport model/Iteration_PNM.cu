#include "hip/hip_runtime.h"
//#include "global.h"

__global__ void deltaCons_calculation(int NY, int *D_CoordMat, int *D_CoordMat_sum, int *D_Prr, double *D_Flux, double *D_deltaCons, double *D_Cons, double *D_Trans)
{
    int tz=blockIdx.x*blockDim.x+threadIdx.x; //becase of the blockSize
    int ty=blockIdx.y*blockDim.y+threadIdx.y;
    int tx=blockIdx.z*blockDim.z+threadIdx.z;
	int tid=tx*NY*NZ+ty*NZ+tz;
    if (tx > NX || ty > NY || tz > NZ || tx < 0 || ty < 0 || tz < 0 || tid >= PE || tid < TP)//!!!!
    {
        return;
    }
	int i, cterm, csterm, poreterm;
	double fluxterm, DCterm, C1term, C2term;

	DCterm = 0;
	cterm = D_CoordMat[tid];
	csterm = D_CoordMat_sum[tid];
	for (i = 0; i < cterm; i++)
	{
		fluxterm = D_Flux[csterm + i];
		poreterm = D_Prr[csterm + i] - 1;
	    C1term = D_Cons[tid];
	    C2term = D_Cons[poreterm];
		if (fluxterm > 0)  // can do this judge in the data0!!!
		{
		    DCterm = DCterm + fluxterm * C2term;
		}
		if (fluxterm < 0)
		{
		    DCterm = DCterm + fluxterm * C1term;
		}
		DCterm = DCterm + D_Trans[csterm + i] * (C2term - C1term);
	}
	D_deltaCons[tid] = DCterm;
}
__global__ void Mintime_calculation(int NY, double *D_deltaCons, double *D_Volume, double *D_mintime, double *D_Cons)
{
    int tz=blockIdx.x*blockDim.x+threadIdx.x; //becase of the blockSize
    int ty=blockIdx.y*blockDim.y+threadIdx.y;
    int tx=blockIdx.z*blockDim.z+threadIdx.z;
	int tid=tx*NY*NZ+ty*NZ+tz;
    if (tx > NX || ty > NY || tz > NZ || tx < 0 || ty < 0 || tz < 0 || tid >= PE || tid < TP)//!!!!
    {
        D_mintime[tid] = 10000;
        return;
    }
	double DCterm = 0.0;
	double DC = 0.0;
	D_mintime[tid] = 0;
	DCterm = D_deltaCons[tid];
	DC = D_Cons[tid];
	if (D_Volume[tid] > 0.0)
		DCterm = DCterm / D_Volume[tid];
	if (DCterm > 0 &&  DC < 0.9999) //DC <= 1.0 &&
    {
	    D_mintime[tid] = (1.0 - DC) / DCterm;
	}
    if (DCterm < 0)
	{
	     D_mintime[tid] = -1.0 * DC / DCterm;
	}
	if(D_mintime[tid] < 10e-20)
	{
	    D_mintime[tid] = 1000000;
	}
      
	D_deltaCons[tid] = DCterm;
}

__global__ void C_calculation(int *d_minindex, int NY, double *D_Cons, double *D_deltaCons, double *D_mintime)
{
    int tz=blockIdx.x*blockDim.x+threadIdx.x; //becase of the blockSize
    int ty=blockIdx.y*blockDim.y+threadIdx.y;
    int tx=blockIdx.z*blockDim.z+threadIdx.z;
	int tid=tx*NY*NZ+ty*NZ+tz;
    if (tx > NX || ty > NY || tz > NZ || tx < 0 || ty < 0 || tz < 0 || tid >= PE || tid < TP)//!!!!
    {
        return;
    }
	double DC = 0.0;
	double DCterm = 0.0;
	DC = D_Cons[tid];
	DCterm = D_deltaCons[tid];
	if (DC <= 1.0)
	{
		D_Cons[tid] = D_Cons[tid] + D_mintime[*d_minindex-1] * D_deltaCons[tid];
	}
	if (DC > 1.0 && DCterm < 0.0)
	{
		D_Cons[tid] = D_Cons[tid] + D_mintime[*d_minindex-1] * D_deltaCons[tid];
	}
	/*D_Cons[tid] = D_Cons[tid] + D_mintime[*d_minindex-1] * D_deltaCons[tid];
	if (DC >= 1.0)
	{
		D_Cons[tid] = 1.0;
	}*/
}


void Iteration()
{   
    deltaCons_calculation << <gridSize, blockSize >> >(NY, D_CoordMat, D_CoordMat_sum, D_Prr, D_Flux, D_deltaCons, D_Cons, D_Trans);
	hipDeviceSynchronize();
    hipDeviceSynchronize();

	Mintime_calculation << <gridSize, blockSize >> >(NY, D_deltaCons, D_Volume, D_mintime, D_Cons);
	hipDeviceSynchronize();
    hipDeviceSynchronize();

    hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
	if(hipblasIdamin(handle, PE, D_mintime, 1, d_minindex) != HIPBLAS_STATUS_SUCCESS) {std::cout << ".";}  //convert to maximum value, to avoid the effect of 0
	hipMemcpy(&hminindex, d_minindex, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&hmint, D_mintime + hminindex - 1, sizeof(double), hipMemcpyDeviceToHost);
	//printf("%d\n",hminindex);
	//hipMemcpy(&hppkk, ppkk, sizeof(double), hipMemcpyDeviceToHost);
	hipblasDestroy(handle);
	//if (hppkk <= 0.0000001) {t = Time;}//error
	//al = hppkk / hppk;
	//hipMemcpy(al, &hal, sizeof(double), hipMemcpyHostToDevice);

	C_calculation << <gridSize, blockSize >> >( d_minindex, NY, D_Cons, D_deltaCons, D_mintime);
	hipDeviceSynchronize();
    hipDeviceSynchronize();
}
