//#include "global.h"

hipError_t Init_PNM()       // allocate space in GPU

{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
    // allocate the area in the GPU
    cudaStatus = hipMalloc((void**)&D_Prr, TF * sizeof(int));
	cudaStatus = hipMalloc((void**)&D_Flux, TF * sizeof(double));// nozero * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_Trans, TF * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_Cons, (PE + 3) * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_Volume, PE * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_mintime, PE * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_deltaCons, PE * sizeof(double));
	cudaStatus = hipMalloc((void**)&D_CoordMat, PE * sizeof(int));
    cudaStatus = hipMalloc((void**)&D_CoordMat_sum, PE * sizeof(int));

	hipMalloc(&d_minindex, sizeof(int));

	//copy data from CPU to GPU
	cudaStatus = hipMemcpy(D_Prr, Prr.data(), TF * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_Flux, Flux.data(), TF * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_Trans, Trans.data(), TF * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_Cons, Cons, (PE + 3) * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_Volume, Volume.data(), PE * sizeof(double), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_CoordMat, CoordMat.data(), PE * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(D_CoordMat_sum, CoordMat_sum.data(), PE * sizeof(int), hipMemcpyHostToDevice);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Copy data failed, Intput!");
	}
	//end copy
	return cudaStatus;
}

void initialize()
{

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		//return 1;
	}

	cudaStatus = Init_PNM(); //malloc memory
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "LBM initialization failed!");
		//return 1;
	}

	//printf("finish blocking");
	//P_setup << <TgridSize, blockSize >> >(NZT, D_deltaCons);//, p0
	//hipDeviceSynchronize();
    //hipDeviceSynchronize();
}

//------------------------------------------------------------------------------
